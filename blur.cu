#include <stdio.h>
#include <string.h>
#include <sys/types.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#define TX 16
#define TY 32
#define NPTSX 200
#define NPTSY 200

__global__
void blur(int *R, int *G, int *B, int *Rnew, int *Gnew, int *Bnew, int colsize, int rowsize) {

  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int i = row * colsize + col;

  if (row != 0 && row != (rowsize-1) && col != 0 && col != (colsize-1)){
  					Rnew[i] = (R[(row+1)*colsize+col]+R[(row-1)*colsize+col]+R[row*colsize+col+1]+R[row*colsize+col-1])/4;
  					Gnew[i] = (G[(row+1)*colsize+col]+G[(row-1)*colsize+col]+G[row*colsize+col+1]+G[row*colsize+col-1])/4;
  					Bnew[i] = (B[(row+1)*colsize+col]+B[(row-1)*colsize+col]+B[row*colsize+col+1]+B[row*colsize+col-1])/4;
  				}
  				else if (row == 0 && col != 0 && col != (colsize-1)){
  					Rnew[i] = (R[(row+1)*colsize+col]+R[row*colsize+col+1]+R[row*colsize+col-1])/3;
  					Gnew[i] = (G[(row+1)*colsize+col]+G[row*colsize+col+1]+G[row*colsize+col-1])/3;
  					Bnew[i] = (B[(row+1)*colsize+col]+B[row*colsize+col+1]+B[row*colsize+col-1])/3;
  				}
  				else if (row == (rowsize-1) && col != 0 && col != (colsize-1)){
  					Rnew[i] = (R[(row-1)*colsize+col]+R[row*colsize+col+1]+R[row*colsize+col-1])/3;
  					Gnew[i] = (G[(row-1)*colsize+col]+G[row*colsize+col+1]+G[row*colsize+col-1])/3;
  					Bnew[i] = (B[(row-1)*colsize+col]+B[row*colsize+col+1]+B[row*colsize+col-1])/3;
  				}
  				else if (col == 0 && row != 0 && row != (rowsize-1)){
  					Rnew[i] = (R[(row+1)*colsize+col]+R[(row-1)*colsize+col]+R[row*colsize+col+1])/3;
  					Gnew[i] = (G[(row+1)*colsize+col]+G[(row-1)*colsize+col]+G[row*colsize+col+1])/3;
  					Bnew[i] = (B[(row+1)*colsize+col]+B[(row-1)*colsize+col]+B[row*colsize+col+1])/3;
  				}
  				else if (col == (colsize-1) && row != 0 && row != (rowsize-1)){
  					Rnew[i] = (R[(row+1)*colsize+col]+R[(row-1)*colsize+col]+R[row*colsize+col-1])/3;
  					Gnew[i] = (G[(row+1)*colsize+col]+G[(row-1)*colsize+col]+G[row*colsize+col-1])/3;
  					Bnew[i] = (B[(row+1)*colsize+col]+B[(row-1)*colsize+col]+B[row*colsize+col-1])/3;
  				}
  				else if (row==0 &&col==0){
  					Rnew[i] = (R[row*colsize+col+1]+R[(row+1)*colsize+col])/2;
  					Gnew[i] = (G[row*colsize+col+1]+G[(row+1)*colsize+col])/2;
  					Bnew[i] = (B[row*colsize+col+1]+B[(row+1)*colsize+col])/2;
  				}
  				else if (row==0 &&col==(colsize-1)){
  					Rnew[i] = (R[row*colsize+col-1]+R[(row+1)*colsize+col])/2;
  					Gnew[i] = (G[row*colsize+col-1]+G[(row+1)*colsize+col])/2;
  					Bnew[i] = (B[row*colsize+col-1]+B[(row+1)*colsize+col])/2;
  				}
  				else if (row==(rowsize-1) &&col==0){
  					Rnew[i] = (R[row*colsize+col+1]+R[(row-1)*colsize+col])/2;
  					Gnew[i] = (G[row*colsize+col+1]+G[(row-1)*colsize+col])/2;
  					Bnew[i] = (B[row*colsize+col+1]+B[(row-1)*colsize+col])/2;
  				}
  				else if (row==(rowsize-1) &&col==(colsize-1)){
  					Rnew[i] = (R[row*colsize+col-1]+R[(row-1)*colsize+col])/2;
  					Gnew[i] = (G[row*colsize+col-1]+G[(row-1)*colsize+col])/2;
  					Bnew[i] = (B[row*colsize+col-1]+B[(row-1)*colsize+col])/2;
  				}

  R[row * colsize + col] = Rnew[row * colsize + col];
  G[row * colsize + col] = Gnew[row * colsize + col];
  B[row * colsize + col] = Bnew[row * colsize + col];
}

int main (int argc, const char * argv[]) {

	static int const maxlen = 200, rowsize = 512, colsize = 428, linelen = 12;
	char str[maxlen], lines[5][maxlen];
	FILE *fp, *fout;
	int nlines = 0;
	unsigned int h1, h2, h3;
	char *sptr;
	int R[rowsize][colsize], G[rowsize][colsize], B[rowsize][colsize];
  int Rnew[(rowsize*colsize)], Gnew[(rowsize*colsize)], Bnew[(rowsize*colsize)];
  int row = 0, col = 0, nblurs, lineno=0, k;
	struct timeval tim;

  gettimeofday(&tim, NULL);
  double inputt1=tim.tv_sec+(tim.tv_usec/1000000.0);
	fp = fopen("David.ps", "r");

	while(! feof(fp))
	{
		fscanf(fp, "\n%[^\n]", str);
		if (nlines < 5) {strcpy((char *)lines[nlines++],(char *)str);}
		else {
			for (sptr=&str[0];*sptr != '\0';sptr+=6){
				sscanf(sptr,"%2x",&h1);
				sscanf(sptr+2,"%2x",&h2);
				sscanf(sptr+4,"%2x",&h3);

				if (col==colsize){
					col = 0;
					row++;
				}
				if (row < rowsize) {
					R[row][col] = h1;
					G[row][col] = h2;
					B[row][col] = h3;
				}
				col++;
			}
		}
	}
	fclose(fp);
  gettimeofday(&tim, NULL);
  double inputt2=tim.tv_sec+(tim.tv_usec/1000000.0);
  printf("Time to read in input file: %f\n",(inputt2-inputt1));


	nblurs = 160;
  int *d_R, *d_G, *d_B, *d_Rnew, *d_Gnew, *d_Bnew;
	int size = sizeof(int)*colsize*rowsize;

  /*printf("Number of blurs: %d\n\n", nblurs);*/

  gettimeofday(&tim, NULL);
  double allocatet1=tim.tv_sec+(tim.tv_usec/1000000.0);

	hipMalloc((void **) &d_R, size);
	hipMalloc((void **) &d_G, size);
	hipMalloc((void **) &d_B, size);
	hipMalloc((void **) &d_Rnew, size);
	hipMalloc((void **) &d_Gnew, size);
	hipMalloc((void **) &d_Bnew, size);

  gettimeofday(&tim, NULL);
  double allocatet2=tim.tv_sec+(tim.tv_usec/1000000.0);
  printf("Time to allocate memory: %f\n", (allocatet2-allocatet1));


  gettimeofday(&tim, NULL);
  double datat1=tim.tv_sec+(tim.tv_usec/1000000.0);

  hipMemcpy(d_R, &R, size, hipMemcpyHostToDevice);
	hipMemcpy(d_G, &G, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, &B, size, hipMemcpyHostToDevice);

  gettimeofday(&tim, NULL);
  double datat2=tim.tv_sec+(tim.tv_usec/1000000.0);
  //printf("Time to transfer data: %f\n", (datat2-datat1));

  dim3 dimGrid(ceil(colsize/(float)TX), ceil(rowsize/(float)TY), 1);
  dim3 dimBlock(TX, TY, 1);

	gettimeofday(&tim, NULL);
	double blurt1=tim.tv_sec+(tim.tv_usec/1000000.0);

  for (int j = 0; j <nblurs; j++){
    blur<<<dimBlock, dimGrid>>>(d_R, d_G, d_B, d_Rnew, d_Gnew, d_Bnew, colsize, rowsize);
  }

  gettimeofday(&tim, NULL);
	double blurt2=tim.tv_sec+(tim.tv_usec/1000000.0);
  printf("Time to blur image: %f\n", (blurt2-blurt1));

  gettimeofday(&tim, NULL);
  double datat3=tim.tv_sec+(tim.tv_usec/1000000.0);
  hipMemcpy(&Rnew, d_Rnew, size, hipMemcpyDeviceToHost);
  hipMemcpy(&Gnew, d_Gnew, size, hipMemcpyDeviceToHost);
  hipMemcpy(&Bnew, d_Bnew, size, hipMemcpyDeviceToHost);
  gettimeofday(&tim, NULL);
  double datat4=tim.tv_sec+(tim.tv_usec/1000000.0);
  printf("Time to transfer data: %f\n", ((datat2-datat1) + (datat4-datat3)));

  gettimeofday(&tim, NULL);
	double outputt1=tim.tv_sec+(tim.tv_usec/1000000.0);
  for (int row=0;row<rowsize;row++){
    for (int col=0;col<colsize;col++){
      int count1 = row*colsize+col;
      R[row][col] = Rnew[count1];
      G[row][col] = Gnew[count1];
      B[row][col] = Bnew[count1];
    }
  }


  hipFree(d_R); hipFree(d_G); hipFree(d_B); hipFree(d_Rnew); hipFree(d_Gnew); hipFree(d_Bnew);

	fout= fopen("DavidBlur.ps", "w");
	for (k=0;k<nlines;k++) fprintf(fout,"\n%s", lines[k]);
	fprintf(fout,"\n");
	for(row=0;row<rowsize;row++){
		for (col=0;col<colsize;col++){
			fprintf(fout,"%02x%02x%02x",R[row][col],G[row][col],B[row][col]);
			lineno++;
			if (lineno==linelen){
				fprintf(fout,"\n");
				lineno = 0;
			}
		}
	}
	fclose(fout);

  gettimeofday(&tim, NULL);
  double outputt2=tim.tv_sec+(tim.tv_usec/1000000.0);
  printf("Time to output image: %f\n", (outputt2-outputt1));
    return 0;
}
